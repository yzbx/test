#include "hip/hip_runtime.h"
#include <stdlib.h>

//#include <npp.h>
//#include <nppi.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>

#define cimg_display 1
#include <CImg.h>

#include "LBPGPU.cuh"

__global__ void calcLBPKernel( const unsigned char * pSrc, unsigned char * pDst, const int width,
			const int height, const LBPMapping * mapping ) {
	// Get the index of the current core
	int tidX = blockIdx.x * blockDim.x + threadIdx.x;
	int tidY = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = tidY * width + tidX;
	int idxX = idx / width;
	int idxY = idx % width;

	// Only continue if a distance 'radius' from the edge
	float r = mapping->radius;
	if( idxX < r || idxX >= width - r || idxY < r || idxY >= height - r )
		return;
//	if( idx != 7 )
//		return;

	const int samples = 8; //mapping->samples;
	float spoints[samples][2];
	float a = 2.f * M_PI / samples;
	for( int i = 0; i < samples; i++ ) {
		spoints[i][0] = +r * cos( float( i * a ) );
		spoints[i][1] = -r * sin( float( i * a ) );
	}

	float val=0;
	int idxC = idx;
	float valC = pSrc[idxC];
	for( int i = 0; i < samples; i++ ) {
		float x = spoints[i][0];
		float y = spoints[i][1];
		int ry = round( y );
		int rx = round( x );
		int indV = idxC + width * ry + rx;
		float valV = pSrc[indV];

		// Check if interpolation is needed.
		if( (fabs( x - rx ) > 1e-6) || (fabs( y - ry ) > 1e-6) ) {
			int fy = floor( y );
			int cy = ceil( y );
			int fx = floor( x );
			int cx = ceil( x );

			int idxV1 = idxC + width * fy + fx;
			int idxV2 = idxC + width * fy + cx;
			int idxV3 = idxC + width * cy + fx;
			int idxV4 = idxC + width * cy + cx;
			// Calculate the interpolation weights.
			float tx = x - fx;
			float ty = y - fy;
			float w1 = (1 - tx) * (1 - ty);
			float w2 = tx * (1 - ty);
			float w3 = (1 - tx) * ty;
			float w4 = tx * ty;
			valV = pSrc[idxV1]*w1 + pSrc[idxV2]*w2 + pSrc[idxV3]*w3 + pSrc[idxV4]*w4;
		}
		val += valV - valC >= 0 ? pow( 2.f, (float)i ) : 0;
	}

	pDst[idx] = (unsigned char) val;
//	printf("%d (%d,%d): %f\n", idx, idxX, idxY, val);
}

/**
 *  The Wrapper function
 */
void calcLBPGPU( const unsigned char * h_src, unsigned char * h_dst, const int width, const int height,
			const LBPMapping * mapping ) {

	unsigned char *d_Src = NULL, *d_Dst = NULL;

	hipMalloc( &d_Src, sizeof(char) * height * width );
	hipMalloc( &d_Dst, sizeof(char) * height * width );
	hipMemset( (void *) d_Dst, 0, sizeof(char) * height * width );
	hipMemcpy( d_Src, h_src, sizeof(char) * height * width, hipMemcpyHostToDevice );

	dim3 numThreadsPerBlock, numBlocks;
	numThreadsPerBlock.x = width;
//	numThreadsPerBlock.y 	= height;
	numBlocks.x = height;
//	numBlocks.y				= 1;

	hipEvent_t start, end;
	hipEventCreate( &start );
	hipEventCreate( &end );
	float time;

	cout << "before gpu call" << endl;
	hipEventRecord( start, 0 );
	calcLBPKernel<<< numBlocks, numThreadsPerBlock >>>( d_Src, d_Dst, width, height, mapping );
	hipEventRecord( end, 0 );
	hipEventSynchronize( end );
	hipEventElapsedTime( &time, start, end );
	hipEventDestroy( start );
	hipEventDestroy( end );
	cout << "after gpu sync. Took " << time / 1000 << "s" << endl;

	hipMemcpy( h_dst, d_Dst, sizeof(char) * height * width, hipMemcpyDeviceToHost );
	hipFree( d_Src );
	hipFree( d_Dst );

}

bool cudaAvailable( void ) {
	int cnt;
	hipGetDeviceCount( &cnt );
	cout << "GPU count: " << cnt << endl;
	return cnt > 0;
}

int main( int argc, char ** argv ) {

	if( !cudaAvailable() ) {
		return -1;
	}

	clock_t startTime, endTime;


#if 0
	unsigned char pixels[] = {78, 87, 84, 81, 92, 98,
				 75, 86, 82, 74, 82, 90,
				 77, 87, 85, 76, 74, 80,
				 91, 98, 91, 81, 77, 79,
				 90, 95, 85, 80, 84, 88,
				 91, 91, 83, 79, 86, 90};
	int w = 6, h = 6;
	cimg_library::CImg<unsigned char> image( pixels, w, h );

#else
	cimg_library::CImg<unsigned char> image( argv[1] );
	unsigned char * pixels = image.data();
	int w = image.width();
	int h = image.height();
#endif
	cout << "Image: " << image.pixel_type() << " (" << image.depth() << ") " << image.width() << "x"
				<< image.height() << ", # Channels=" << image.spectrum() << endl;

//	image.display();
//	for( int j = 0; j < image.height(); ++j ) {
//		for( int i = 0; i < image.width(); ++i ) {
//			printf( "%3d ", (int) image.atXY(i,j) );
//		}
//		printf( "\n" );
//	}

	LBPMapping mapping;

	startTime = clock();
	calcLBPGPU( pixels, pixels, w, h, &mapping );
	endTime = clock();
	// hipDeviceReset must be called before exiting in order for profiling and
	    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	    if (cudaStatus != hipSuccess) {
	        fprintf(stderr, "hipDeviceReset failed!");
	        return 1;
	    }
#if 0
	for( int j = 0; j < h; j++ ) {
		for( int i = 0; i < w; ++i ) {
			printf( "%3d ", (int) pixels[j * w + i] );
		}
		printf( "\n" );
	}
#elseif 0
	image.assign( pixels, w, h );
		image.display();
#endif

	cout << "Example took " << double( endTime - startTime ) / double( CLOCKS_PER_SEC ) << "s"
				<< endl;



	return 0;

}
